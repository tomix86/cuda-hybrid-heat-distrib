#include "hip/hip_runtime.h"
#include <cstdio>
#include <sstream>
#include <stdexcept>
#include "kernel.hpp"
#include "Mesh.hpp"

static std::string to_string(hipError_t error) {
	char buf[256];
	snprintf(buf, 256, "%d", error);
	return buf;
}


class CudaError : public std::runtime_error {
public:
	CudaError(std::string source, hipError_t errorCode) :
		std::runtime_error( source + ": code" + to_string(errorCode) + ": " + hipGetErrorString(errorCode) ) {
	}
};

#define checkCudaErrors( val ) checkError( ( val ), #val, __FILE__, __LINE__ )
void checkError(hipError_t result, const char* calledFunc,  const char* file, int line) {
	if (result) {
		std::ostringstream ss;
		ss << file << ": " << line << " {" << calledFunc << '}';

		throw CudaError(ss.str(), result);
	}
}

__global__ void meshUpdateKernel(float* mesh_in, float* mesh_out, size_t pitch, unsigned size) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if ( x > 0 && x < size - 1 && y > 0 && y < size - 1) {
		const float t_left = *getElem(mesh_in, pitch, y, x - 1);
		const float t_right = *getElem(mesh_in, pitch, y, x + 1);
		const float t_top = *getElem(mesh_in, pitch, y - 1, x); 
		const float t_bottom = *getElem(mesh_in, pitch, y + 1, x);

		const float newTemperature = (t_left + t_right + t_top + t_bottom) / 4;
		
		*getElem(mesh_out, pitch, y, x) = newTemperature;
	}
}


// optimal block size is 128,1,1
__global__ void meshUpdateKernel_opt1(float *mesh_in, float *mesh_out, size_t pitch, unsigned size) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	//TODO: switch to dynamic shared memory
	__shared__ float shared[3][128 + 2];
/*
	if (threadIdx.x == 0) {
		if (x > 0) {
			shared[1][0] = *getElem(mesh_in, pitch, y, x - 1);
		}
		else {
			shared[1][1] = *getElem(mesh_in, pitch, y, x);
		}
	}

	if (threadIdx.x == blockDim.x - 1) {
		if (x < size - 1) {
			shared[1][blockDim.x + 1] = *getElem(mesh_in, pitch, y, x + 1);
		}
		else {
			const auto pos = size - blockIdx.x * blockDim.x;
			shared[1][pos] = *getElem(mesh_in, pitch, y, blockIdx.x * blockDim.x + pos - 1);
		}
	}*/
	
	if (x > 0 && x < size - 1 && y > 0 && y < size - 1) {
//		shared[1][threadIdx.x + 1 - 1] = *getElem(mesh_in, pitch, y, x - 1);
//		shared[1][threadIdx.x + 1 + 1] = *getElem(mesh_in, pitch, y, x + 1);
		shared[0][threadIdx.x + 1] = *getElem(mesh_in, pitch, y-1, x);
		shared[1][threadIdx.x + 1] = *getElem(mesh_in, pitch, y, x);
		shared[2][threadIdx.x + 1] = *getElem(mesh_in, pitch, y+1, x);

		__syncthreads();

		const float t_l = shared[1][threadIdx.x + 1 - 1];
		const float t_r = shared[1][threadIdx.x + 1 + 1];
		const float t_t = shared[0][threadIdx.x + 1];
		const float t_b = shared[2][threadIdx.x + 1];

		const float newTemperature = (t_l + t_r + t_b + t_t) / 4;

//		printf("[%d,%d]: {%f;%f;%f;%f}: %f\n", x, y, t_l, t_r, t_t, t_b, newTemperature);

		*getElem(mesh_out, pitch, y, x) = newTemperature;
	}
}


void cuda() {
	size_t pitch;
	float *temperature = allocMeshLinear(pitch);
	size_t d_pitch;
	float *d_temperature_in, *d_temperature_out;

	try {
		checkCudaErrors(hipMallocPitch(&d_temperature_in, &d_pitch, MESH_SIZE_EXTENDED * sizeof(float), MESH_SIZE_EXTENDED));
		checkCudaErrors(hipMallocPitch(&d_temperature_out, &d_pitch, MESH_SIZE_EXTENDED * sizeof(float), MESH_SIZE_EXTENDED));
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
		return;
	}

	try {
		SimpleTimer t( "CUDA implementation" );
		dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
		unsigned computedGridDimX = (MESH_SIZE_EXTENDED + blockSize.x - 1) / blockSize.x;
		unsigned computedGridDimY = (MESH_SIZE_EXTENDED + blockSize.y - 1) / blockSize.y;
		dim3 gridSize(computedGridDimX, computedGridDimY);

		checkCudaErrors(hipMemcpy2D(d_temperature_in, d_pitch, temperature, pitch, MESH_SIZE_EXTENDED * sizeof(float), MESH_SIZE_EXTENDED, hipMemcpyHostToDevice));
		checkCudaErrors(hipMemcpy2D(d_temperature_out, d_pitch, d_temperature_in, d_pitch, MESH_SIZE_EXTENDED * sizeof(float), MESH_SIZE_EXTENDED, hipMemcpyDeviceToDevice));

		for (int step = 0; step < STEPS; ++step) {
			meshUpdateKernel << < gridSize, blockSize >> > (d_temperature_in, d_temperature_out, d_pitch, MESH_SIZE_EXTENDED);
			checkCudaErrors(hipGetLastError()); // Check for any errors launching the kernel
			checkCudaErrors(hipDeviceSynchronize());// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.
			std::swap(d_temperature_in, d_temperature_out);
		}

		checkCudaErrors(hipMemcpy2D(temperature, pitch, d_temperature_in, d_pitch, MESH_SIZE_EXTENDED * sizeof(float), MESH_SIZE_EXTENDED, hipMemcpyDeviceToHost));
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
	}

	validateResults(temperature, pitch);

	delete[] temperature;
	try {
		checkCudaErrors(hipFree(d_temperature_in));
		checkCudaErrors(hipFree(d_temperature_out));
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
	}

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	checkCudaErrors(hipDeviceReset());
}

//
//
// Hybrid implementation
//
//

__global__ void meshUpdateKernel_hybrid(float* mesh_in, float* mesh_out, size_t pitch, unsigned size_x, unsigned size_y) {
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x > 0 && x < size_x - 1 && y > 0 && y < size_y - 1) {
		const float t_left = *getElem(mesh_in, pitch, y, x - 1);
		const float t_right = *getElem(mesh_in, pitch, y, x + 1);
		const float t_top = *getElem(mesh_in, pitch, y - 1, x);
		const float t_bottom = *getElem(mesh_in, pitch, y + 1, x);

		const float newTemperature = (t_left + t_right + t_top + t_bottom) / 4;

		*getElem(mesh_out, pitch, y, x) = newTemperature;

	//	printf("[%d,%d]: {%f;%f;%f;%f}: %f\n", x, y, t_left, t_right, t_top, t_bottom, newTemperature);
	}
}


HybridCuda::HybridCuda(size_t divisionPoint, size_t pitch, int deviceId) :
DIVISION_POINT(divisionPoint),
pitch(pitch),
deviceId(deviceId) {
	part = (deviceId == 0 ? BOTTOM : TOP);

	if (part == BOTTOM) {
		allocNumRows = MESH_SIZE_EXTENDED - (DIVISION_POINT - 1);
	}
	else {
		allocNumRows = DIVISION_POINT + 1;
	}

	setDevice();

	try {
		checkCudaErrors(hipMallocPitch(&d_temperature_in, &d_pitch, MESH_SIZE_EXTENDED * sizeof(float), allocNumRows));
		checkCudaErrors(hipMallocPitch(&d_temperature_out, &d_pitch, MESH_SIZE_EXTENDED * sizeof(float), allocNumRows));
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
		return;
	}
}

HybridCuda::~HybridCuda() {
	setDevice();

	try {
		checkCudaErrors(hipFree(d_temperature_in));
		checkCudaErrors(hipFree(d_temperature_out));
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
	}
}

//TODO: overlap computation and communication - launch separate streams
void HybridCuda::launchCompute(float* temperature_in) {
	setDevice();

	try {
		dim3 blockSize(BLOCK_DIM_X, BLOCK_DIM_Y);
		unsigned computedGridDimX = (MESH_SIZE_EXTENDED + blockSize.x - 1) / blockSize.x;
		unsigned computedGridDimY = (allocNumRows + blockSize.y - 1) / blockSize.y;
		dim3 gridSize(computedGridDimX, computedGridDimY);

		float* srcPtr;
		if (part == BOTTOM) {
			srcPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(temperature_in) + (DIVISION_POINT - 1) * pitch);
		}
		else {
			srcPtr = temperature_in;
		}

		checkCudaErrors(hipMemcpy2D(d_temperature_in, d_pitch, srcPtr, pitch, MESH_SIZE_EXTENDED * sizeof(float), 1, hipMemcpyHostToDevice));

		meshUpdateKernel_hybrid<<< gridSize, blockSize >>> (d_temperature_in, d_temperature_out, d_pitch, MESH_SIZE_EXTENDED, allocNumRows);
	//	checkCudaErrors(hipGetLastError()); // Check for any errors launching the kernel
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
	}
}

void HybridCuda::finalizeCompute(float* temperature_out) {
	setDevice();

	try {
		checkCudaErrors(hipDeviceSynchronize());// hipDeviceSynchronize waits for the kernel to finish, and returns any errors encountered during the launch.

		float *srcPtr, *dstPtr;
		if (part == BOTTOM) {
			dstPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(temperature_out) + DIVISION_POINT * pitch);
			srcPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(d_temperature_out) + d_pitch);
		}
		else {
			dstPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(temperature_out) + (DIVISION_POINT + 1) * pitch);
			srcPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(d_temperature_out) + (allocNumRows - 1) * pitch); //TODO: verify
		}


		checkCudaErrors(hipMemcpy2D(dstPtr, pitch, srcPtr, d_pitch, MESH_SIZE_EXTENDED * sizeof(float), 1, hipMemcpyDeviceToHost));

		std::swap(d_temperature_in, d_temperature_out);
	}
	catch (CudaError& err) {
		std::cout << err.what() << std::endl;
	}
}

void HybridCuda::copyInitial(float* temperature_in) {
	setDevice();

	float* srcPtr;
	if (part == BOTTOM) {
		srcPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(temperature_in) + (DIVISION_POINT - 1) * pitch);
	}
	else {
		srcPtr = temperature_in;
	}
	checkCudaErrors(hipMemcpy2D(d_temperature_in, d_pitch, srcPtr, pitch, MESH_SIZE_EXTENDED * sizeof(float), allocNumRows, hipMemcpyHostToDevice));

	//TODO: remove this copy - copy only last row
	checkCudaErrors(hipMemcpy2D(d_temperature_out, d_pitch, d_temperature_in, d_pitch, MESH_SIZE_EXTENDED * sizeof(float), allocNumRows, hipMemcpyDeviceToDevice));
}

void HybridCuda::copyFinal(float* temperature_out) {
	setDevice();

	float *srcPtr, *dstPtr;
	if (part == BOTTOM) {
		dstPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(temperature_out) + DIVISION_POINT * pitch);
		srcPtr = reinterpret_cast<float*>(reinterpret_cast<char*>(d_temperature_in) + d_pitch);
	}
	else {
		dstPtr = temperature_out;
		srcPtr = d_temperature_in;
	}

	checkCudaErrors(hipMemcpy2D(dstPtr, pitch, srcPtr, d_pitch, MESH_SIZE_EXTENDED * sizeof(float), allocNumRows - 1, hipMemcpyDeviceToHost));
}

void HybridCuda::setDevice() {
	hipSetDevice(deviceId);
}
